
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>


__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}



void cython_add(int *a, int *b, int *result, int N) {
	int size = N*sizeof(int);
	int *d_a, *d_b, *d_result;
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_result, size);
	// Copy array a to d_a, b to d_b on the device.
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_result, size);
	add<<<N,1>>>(d_a, d_b, d_result);
	hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_result);

}