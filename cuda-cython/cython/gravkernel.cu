
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void __global__ fcuda(double *f, double *r, double *q, int N) {
    int i = blockIdx.x;
    double xi = r[2*i + 0];
    double yi = r[2*i + 1];
    double dx, dy, Fval, theta;
    f[i] = 0;
    for(int j = 0; j < N; j++) {
	if (i != j) {
        dx = xi - r[2*j+0];
        dy = yi - r[2*j+1];
        theta = atan2(dy, dx);
        Fval = q[i]*q[j]/(4*M_PI*8.854e-12*(dx*dx + dy*dy));
        f[2*i+0] += Fval*cos(theta);
        f[2*i+1] += Fval*sin(theta);
       }
   }
}

