#include "gravkernel.cu"
#include <grav.hh>
using namespace std;

void F(double *f, double *r, double *q, int N) {
    double *d_f, *d_r, *d_q;
    int size = sizeof(double) * N;
    hipMalloc((void **) &d_f, size);
    hipMalloc((void **) &d_r, size);
    hipMalloc((void **) &d_q, size);
    hipMemcpy(d_r, r, size, hipMemcpyHostToDevice);
    hipMemcpy(d_q, q, size, hipMemcpyHostToDevice);
    fcuda<<<N,1>>>(d_f, d_r, d_q, N);
    hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost);
    hipFree(d_r);
    hipFree(d_q);
    hipFree(d_f);
}
